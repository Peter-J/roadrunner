#include "hip/hip_runtime.h"
#include "GPUSimIntegratorInt.h"

#include <stdlib.h>
#include <stdio.h>

using namespace rr::rrgpu;

typedef float RKReal;

#define RK4BLOCKS 4

#define RK_COEF_LEN RK4BLOCKS*RK4BLOCKS*n
#define RK_COEF_OFFSET(gen, idx, component) gen*RK4BLOCKS*n + idx*n + component

// #define RK_STATE_VEC_LEN RK4BLOCKS*n*sizeof(RKReal) // FIXME?
#define RK_STATE_VEC_LEN RK4BLOCKS*n
#define RK_STATE_VEC_OFFSET(idx, component) idx*n + component

#define RK_TIME_VEC_LEN RK4BLOCKS

/**
 * @author JKM
 * @brief RK4 kernel
 * @param[in] n The size of the state vector
 * @param[in] y The state vector
 */
__global__ void kern(int n,  RKReal h, GPUSimIntegratorInt& intf) {
    extern __shared__ RKReal k[];
    RKReal* f = &k[RK4BLOCKS*RK4BLOCKS*n];
    RKReal* t = &f[RK_STATE_VEC_LEN];

//     RKReal t = t0;

    printf("kern\n");
    // hope you can render unicode
    // y ∈ ℝ(n)
    // so, to access k use pattern
    // offset = generation*RK4BLOCKS*n + block*n + i
    // where i is the index of the ith component in ℝ(n)
    // and block corresponds to the index of the coefficient k1..4

    // initialize k
    for (int j=0; j<RK4BLOCKS; ++j) {
        k[RK_COEF_OFFSET(j, blockIdx.x, threadIdx.x)] = 0;
        printf("k[RK_COEF_OFFSET(%d, %d, %d)] = %f\n", j, blockIdx.x, threadIdx.x, k[RK_COEF_OFFSET(j, blockIdx.x, threadIdx.x)]);
//         printf("k[%d*%d*%d + %d*%d + %d] = %f\n", j, RK4BLOCKS, n, blockIdx.x, n, threadIdx.x, k[j*RK4BLOCKS*n + blockIdx.x*n + threadIdx.x]);
//         printf("k[%d*%d*%d + %d*%d + %d], offset = %d\n", j, RK4BLOCKS, n, blockIdx.x, n, threadIdx.x, j*RK4BLOCKS*n + blockIdx.x*n + threadIdx.x);
    }
//
//     // initialize state vector
    f[blockIdx.x*n + threadIdx.x] = 0;
    printf("f[%d*%d + %d] = %f\n", blockIdx.x, n, threadIdx.x, f[blockIdx.x*n + threadIdx.x]);

    // initialize time vector
    t[0] = 0;
    t[1] = 0.5*h; // 0.5f?
    t[2] = 0.5*h; // 0.5f?
    t[3] = h;

    // current generation
//     int m=0;

    // model evaluation

}

// size of the shared mem blocks plush padding where applicable
#define RK_COEF_SIZE RK_COEF_LEN*sizeof(RKReal)
#define RK_STATE_VEC_SIZE RK_STATE_VEC_LEN*sizeof(RKReal)
#define RK_TIME_VEC_SIZE RK_TIME_VEC_LEN*sizeof(RKReal)

void GPUIntMESerial(GPUSimIntegratorInt& intf) {
    int n = intf.getStateVectorSize();

    printf("launchKern state vec size %d\n", n);

//     RKReal* y;
//     hipMalloc(&y, n*sizeof(RKReal));

    printf("launchKern n %d\nRK_COEF_SIZE %d\nRK_STATE_VEC_SIZE %d\nRK_TIME_VEC_SIZE %d\n", n,  RK_COEF_SIZE, RK_STATE_VEC_SIZE, RK_TIME_VEC_SIZE);

    size_t shared_mem_size =
        RK_COEF_SIZE + // RK coefficients
        RK_STATE_VEC_SIZE + // state vector
        RK_TIME_VEC_SIZE // time vector
        ;

    printf("launchKern shared_mem_size = %u\n", shared_mem_size);

    // execution configuration
    // * first param: num blocks (four blocks for RK4)
    // * second param: threads per block
    // * third param: shared memory size
    // shared mem for k's: RK4BLOCKS*RK4BLOCKS*n*sizeof(float)
    // shared mem for state vectors: RK4BLOCKS*n*sizeof(float)
    // set to size of the state vector*4 because it is
    // quadruple buffered (enables concurrent execution)
    kern<<<RK4BLOCKS, n, shared_mem_size>>>(n, 0.1, intf);
//     kern<<<4, n
//         >>>(n, 0.1);

    hipDeviceSynchronize();

//     hipFree(y);
}
